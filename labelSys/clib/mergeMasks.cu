#include "hip/hip_runtime.h"

# include "common.h"
# include <stdlib.h>
# include <time.h>
# include <stdio.h>

typedef unsigned char arr_t;
typedef unsigned char msk_t;
typedef int coord_t;


__global__ void kernel_fuseColor(arr_t * ddest, msk_t * dmsks, arr_t * dcolors, int im_h, int im_w, int n_msks){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= im_h * im_w){
        return;
    }

    arr_t * dst = ddest + i*3;
    msk_t * msk;
    arr_t * color;

    float sum_color[3] = {0, 0, 0};
    int counter = 0;

    for (int m = 0; m<n_msks; m++){
        msk = dmsks + (im_h*im_w)*m + i;
        if (*msk == 1){
            color = dcolors + 3*m;
            sum_color[0] += color[0];
            sum_color[1] += color[1];
            sum_color[2] += color[2];
            counter++;
        }
    }
    if (counter > 0){
        dst[0] = sum_color[0]/counter;
        dst[1] = sum_color[1]/counter;
        dst[2] = sum_color[2]/counter;
    }
}

extern "C"{
/* 
 *
 * dst - destination color mask with length im_h*im_w*3
 * msks - src bool masks with length n_msks*im_h*im_w
 * colors - aim colors with length n_msks*3
 *
 * */
EXPORT void mergeBool2Color2D( arr_t * dest, msk_t * msks, arr_t * colors, int im_h, int im_w, int n_msks ){
    arr_t* ddest;
    msk_t* dmsks;
    arr_t* dcolors;

    int nbytes_dest = sizeof(arr_t)*im_h*im_w*3;
    int nbytes_msks = sizeof(msk_t)*n_msks*im_h*im_w;
    int nbytes_colors = sizeof(arr_t)*n_msks*3;

    clock_t start_time, end_time;

    start_time = clock();
    hipMalloc(&ddest, nbytes_dest);
    hipMalloc(&dmsks, nbytes_msks);
    hipMalloc(&dcolors, nbytes_colors);
    end_time = clock();
    // printf("Time for cuda allocating memories: %fs\n", (float)(end_time - start_time)/CLOCKS_PER_SEC);

    start_time = clock();
    hipMemcpy(ddest, dest, nbytes_dest, hipMemcpyHostToDevice);
    hipMemcpy(dmsks, msks, nbytes_msks, hipMemcpyHostToDevice);
    hipMemcpy(dcolors, colors, nbytes_colors, hipMemcpyHostToDevice);
    end_time = clock();
    // printf("Time for cuda copying memories: %fs\n", (float)(end_time - start_time)/CLOCKS_PER_SEC);

    const int block_size = 1024;
    int n_blocks = im_h*im_w/block_size + 1;
    kernel_fuseColor<<<n_blocks, block_size>>>(ddest, dmsks, dcolors, im_h, im_w, n_msks);

    hipDeviceSynchronize();
    hipMemcpy(dest, ddest, nbytes_dest, hipMemcpyDeviceToHost);

    hipFree(ddest);
    hipFree(dmsks);
    hipFree(dcolors);
}
}
